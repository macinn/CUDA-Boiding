#include "hip/hip_runtime.h"
#include "BoidsLogic.cpp"
#include <hip/hip_runtime.h>
#include <random>

#include <cuda_gl_interop.h>
#include ""

#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/gather.h>
#include <thrust/execution_policy.h>
#include <stdexcept>

#define BLOCK_SIZE 1024
#define BLOCK_NUMBER 4

#pragma once

__device__ int clamp(int min, int x, int max)
{
    return x < min ? min : (x > max ? max : x);
}

__device__ float distance2(glm::vec3 a, glm::vec3 b) {
    return (a.x - b.x) * (a.x - b.x)
        + (a.y - b.y) * (a.y - b.y)
        + (a.z - b.z) * (a.z - b.z);
}

__device__ float l2Norm(glm::vec3 a) {
    return sqrtf(a.x * a.x + a.y * a.y + a.z * a.z);
}

__global__ void assignGridIndKernel(double gridSize, int gridSizeX, int gridSizeY, int gridSizeZ,
    uint N, glm::vec3* dev_boids_p, int* dev_boids_grid_ind_1, int* dev_boids_grid_ind_2)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    while (idx < N)
    {
        int x = dev_boids_p[idx].x / gridSize;
        int y = dev_boids_p[idx].y / gridSize;
        int z = dev_boids_p[idx].z / gridSize;

        x = clamp(0, x, gridSizeX - 1);
        y = clamp(0, y, gridSizeY - 1);
        z = clamp(0, z, gridSizeZ - 1);

        dev_boids_grid_ind_1[idx] = x + y * gridSizeX + z * gridSizeX * gridSizeY;
        dev_boids_grid_ind_2[idx] = dev_boids_grid_ind_1[idx];

        idx += BLOCK_SIZE * BLOCK_NUMBER;
    }
}

__global__ void updateBoidsKernel(const float dt, const uint N, 
    glm::vec3* dev_boids_p, glm::vec3* dev_boids_v, 
    const int* dev_boids_grid_ind, const int* dev_grid_start, const int* dev_grid_end,
    const int gridSizeX, const int gridSizeY, const int gridSizeZ,
    const float turnFactor, const float visualRange, const float protectedRange,
    const float centeringFactor, float avoidFactor, float matchingFactor,
    const float maxSpeed, const float minSpeed,
    const uint width, const uint height, const uint depth,
    const float marginFactor)
{
    float visualRangeSquared = visualRange * visualRange;
    float protectedRangeSquared = protectedRange * protectedRange;
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    while (idx < N)
    {
        int current_grid_id = dev_boids_grid_ind[idx];
        uint countVisible = 0;
        uint countClose = 0;
        glm::vec3 vel = glm::vec3(0.0f);
        glm::vec3 center = glm::vec3(0.0f);
        glm::vec3 close = glm::vec3(0.0f);

        int ind_x = current_grid_id % gridSizeX;
        int ind_y = (current_grid_id / gridSizeX) % gridSizeY;
        int ind_z = current_grid_id / (gridSizeX * gridSizeY);

        for (int i_x = -(ind_x > 0); i_x <= (ind_x < gridSizeX); i_x++)
            for (int i_y = -(ind_y > 0); i_y <= (ind_y < gridSizeY); i_y++)
                for (int i_z = -(ind_z > 0); i_z <= (ind_z < gridSizeZ); i_z++)
                {
                    int neighbour_grid_id =
                        +i_x
                        + i_y * gridSizeX
                        + i_z * gridSizeX * gridSizeY;

                    for (int j = dev_grid_start[neighbour_grid_id]; j <= dev_grid_end[neighbour_grid_id]; j++)
                    {
                        if (idx != j) {
                            float distanceSquared = distance2(dev_boids_p[idx], dev_boids_p[j]);
                            if (distanceSquared < visualRangeSquared)
                            {
                                center += dev_boids_p[j];
                                countVisible++;

                                if (distanceSquared < protectedRangeSquared)
                                {
                                    vel += dev_boids_v[j];
                                    close -= dev_boids_p[j];
                                    countClose++;
                                }
                            }

                        }
                    }

                }

        if (countVisible > 0) {
            center /= countVisible;

            if (countClose > 0) {
                vel /= countClose;
            }
        }

        close += (float)countClose * dev_boids_p[idx];
        dev_boids_v[idx] +=
            (center - dev_boids_p[idx]) * centeringFactor	// cohesion
            + close * avoidFactor						// separation	
            + (vel - dev_boids_v[idx]) * matchingFactor;		// alignment

        if (dev_boids_p[idx].x < width * marginFactor) {
            dev_boids_v[idx].x += turnFactor;
        }
        if (dev_boids_p[idx].x > width * (1 - marginFactor)) {
            dev_boids_v[idx].x -= turnFactor;
        }
        if (dev_boids_p[idx].y < height * marginFactor) {
            dev_boids_v[idx].y += turnFactor;
        }
        if (dev_boids_p[idx].y > height * (1 - marginFactor)) {
            dev_boids_v[idx].y -= turnFactor;
        }
        if (dev_boids_p[idx].z < depth * marginFactor) {
            dev_boids_v[idx].z += turnFactor;
        }
        if (dev_boids_p[idx].z > depth * (1 - marginFactor)) {
            dev_boids_v[idx].z -= turnFactor;
        }
    
        float speed = l2Norm(dev_boids_v[idx]);
        if (speed > maxSpeed) {
            dev_boids_v[idx] /= speed;
            dev_boids_v[idx] *= maxSpeed;
        }
        else if (speed < minSpeed) {
            dev_boids_v[idx] /= speed;
            dev_boids_v[idx] *= minSpeed;
        }

        dev_boids_p[idx] += dev_boids_v[idx] * dt;

        idx += BLOCK_SIZE * BLOCK_NUMBER;
    }
}

__global__ void findGridStartEnd(int* dev_grid_start, int* dev_grid_end, int* dev_boids_grid_ind, int gridCount, uint N)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < gridCount)
    {
        dev_grid_start[idx] = gridCount;
        dev_grid_end[idx] = -1;
    }
    while (idx < N)
    {
        if (idx == 0) {
            dev_grid_start[dev_boids_grid_ind[idx]] = 0;
        }
        else if (dev_boids_grid_ind[idx] != dev_boids_grid_ind[idx - 1])
        {
            dev_grid_end[dev_boids_grid_ind[idx - 1]] = idx - 1;
            dev_grid_start[dev_boids_grid_ind[idx]] = idx;

            if (idx == N - 1)
            {
                dev_grid_end[dev_boids_grid_ind[idx]] = idx;
            }
        }

        idx += BLOCK_SIZE * BLOCK_NUMBER;
    }
}

class BoidsLogicGPU: public BoidsLogic {
private:
    glm::vec3* dev_boids_p;
    glm::vec3* dev_boids_v;
    hipGraphicsResource* cuda_boids_p;
    hipGraphicsResource* cuda_boids_v;
    int* dev_boids_grid_ind_1;
    int* dev_boids_grid_ind_2;
    int* dev_grid_start;
    int* dev_grid_end;
    double gridSize;
    bool firstRun = true;


    void init()
    {
        hipError_t cudaStatus;

        cudaStatus = hipMemset(dev_boids_v, 0, N * sizeof(glm::vec3));
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("hipMemset failed!");
        }

        glm::vec3* boids_p = new glm::vec3[N]();
        std::default_random_engine rd{ static_cast<long uint>(time(0)) };
        std::mt19937 gen{ rd() };
        std::uniform_real_distribution<> w(0, width);
        std::uniform_real_distribution<> h(0, height);
        std::uniform_real_distribution<> z(0, depth);

        for (uint i = 0; i < N; i++) {
            boids_p[i] = glm::vec3(w(gen), h(gen), z(gen));
        }

        cudaStatus = hipMemcpy(dev_boids_v, boids_p, N * sizeof(glm::vec3), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("hipMemcpy failed!");
        }

        delete[] boids_p;
    }

    void updateData(float dt)
    {
        hipError_t cudaStatus;

        int gridSizeX = (width - 1) / gridSize + 1;
        int gridSizeY = (height - 1) / gridSize + 1;
        int gridSizeZ = (depth - 1) / gridSize + 1;

        updateBoidsKernel << < BLOCK_NUMBER, BLOCK_SIZE >> > (
             dt,  N,
             dev_boids_p,  dev_boids_v,
             dev_boids_grid_ind_1,  dev_grid_start,  dev_grid_end,
             gridSizeX,  gridSizeY, gridSizeZ,
             turnFactor,  visualRange,  protectedRange,
             centeringFactor,  avoidFactor,  matchingFactor,
             maxSpeed,  minSpeed,
             width,  height,  depth,
             marginFactor);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("updateBoidsKernel failed!");
        }

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("hipDeviceSynchronize returned error after launching updateBoidsKernel!");
        }
    }

    void updateBuffers(GLuint positionBuffer, GLuint velocityBuffer)
    {

    }

    void assignGridInd()
    {
        hipError_t cudaStatus;

        int gridSizeX = (width - 1) / gridSize + 1;
        int gridSizeY = (height - 1) / gridSize + 1;
        int gridSizeZ = (depth - 1) / gridSize + 1;

        assignGridIndKernel << < BLOCK_NUMBER, BLOCK_SIZE >> > (this->gridSize, gridSizeX, gridSizeY, gridSizeZ, this->N, this->dev_boids_p,
            dev_boids_grid_ind_1, dev_boids_grid_ind_2);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("assignGridIndKernel failed!");
        }

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("hipDeviceSynchronize returned error after launching assignGridIndKernel!");
        }
    }

    void sortGrid()
    {
        thrust::sort_by_key(thrust::device, dev_boids_grid_ind_1, dev_boids_grid_ind_1 + N, dev_boids_v);
        thrust::sort_by_key(thrust::device, dev_boids_grid_ind_2, dev_boids_grid_ind_2 + N, dev_boids_p);

        hipError_t cudaStatus;

        int gridSizeX = (width - 1) / gridSize + 1;
        int gridSizeY = (height - 1) / gridSize + 1;
        int gridSizeZ = (depth - 1) / gridSize + 1;
        
        findGridStartEnd << < BLOCK_NUMBER, BLOCK_SIZE >> > (dev_grid_start, dev_grid_end, dev_boids_grid_ind_1, gridSizeX * gridSizeY * gridSizeZ, N);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("findGridStartEnd failed!");
        }

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("hipDeviceSynchronize returned error after launching findGridStartEnd!");
        }
    }

public:
	BoidsLogicGPU(uint N, uint width, uint height, uint depth) :
        BoidsLogic(N, width, height, depth)
	{
        hipError_t cudaStatus;

        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("hipSetDevice failed!");
        }

        cudaStatus = hipMalloc((void**)&dev_boids_v, N * sizeof(glm::vec3));
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("hipMalloc failed!");
        }

        cudaStatus = hipMalloc((void**)&dev_boids_p, N * sizeof(glm::vec3));
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("hipMalloc failed!");
        }

        cudaStatus = hipMalloc((void**)&dev_boids_grid_ind_1, N * sizeof(int));
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("hipMalloc failed!");
        }

        cudaStatus = hipMalloc((void**)&dev_boids_grid_ind_2, N * sizeof(int));
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("hipMalloc failed!");
        }

        cudaStatus = hipMalloc((void**)&dev_grid_start, N * sizeof(int));
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("hipMalloc failed!");
        }

        cudaStatus = hipMalloc((void**)&dev_grid_end, N * sizeof(int));
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("hipMalloc failed!");
        }
        gridSize = 2 * visualRange;



        
        // populate with random values
        this->init();
	}
    ~BoidsLogicGPU() {
        hipFree(dev_boids_v);
        hipFree(dev_boids_p);
        hipFree(dev_boids_grid_ind_1);
        hipFree(dev_boids_grid_ind_2);
        hipFree(dev_grid_start);
        hipFree(dev_grid_end);
    }

    // Update boids position and velocity
    void update(float dt, GLuint positionBuffer, GLuint velocityBuffer) {
        hipError_t cudaStatus;
        if (firstRun)
        {
            firstRun = false;
            cudaStatus = hipGraphicsGLRegisterBuffer(&cuda_boids_p, positionBuffer, hipGraphicsRegisterFlagsWriteDiscard);
            if (cudaStatus != hipSuccess) {
                throw std::runtime_error("hipGraphicsGLRegisterBuffer failed!");
            }


            cudaStatus = hipGraphicsGLRegisterBuffer(&cuda_boids_v, velocityBuffer, hipGraphicsRegisterFlagsWriteDiscard);
            if (cudaStatus != hipSuccess) {
                throw std::runtime_error("hipGraphicsGLRegisterBuffer failed!");
            }

        }
        size_t size;
        assignGridInd();
        sortGrid();

        cudaStatus = hipGraphicsResourceGetMappedPointer((void**)&dev_boids_p, &size, cuda_boids_p);
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("hipGraphicsResourceGetMappedPointer failed!");
        }
        cudaStatus = hipGraphicsResourceGetMappedPointer((void**)&dev_boids_v, &size, cuda_boids_v);
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("hipGraphicsResourceGetMappedPointer failed!");
        }

        updateData(dt);

        cudaStatus = hipGraphicsUnmapResources(1, &cuda_boids_p, 0);
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("hipGraphicsUnmapResources failed!");
        }
        cudaStatus = hipGraphicsUnmapResources(1, &cuda_boids_v, 0);
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("hipGraphicsUnmapResources failed!");
        }
        //updateBuffers(positionBuffer, velocityBuffer);
    }
};

