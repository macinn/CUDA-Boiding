#include <iostream>
#include <hip/hip_runtime.h>

int checkCuda() {
    // Check if CUDA is available
    hipError_t cudaStatus = hipRuntimeGetVersion(nullptr);

    if (cudaStatus == hipSuccess) {
        int deviceCount;
        hipGetDeviceCount(&deviceCount);

        if (deviceCount > 0) {
            std::cout << "CUDA is available on this system." << std::endl;
            std::cout << "Number of CUDA devices: " << deviceCount << std::endl;

            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, 0);

            std::cout << "CUDA device name: " << deviceProp.name << std::endl;
        }
        else {
            std::cout << "No CUDA devices found on this system." << std::endl;
        }
    }
    else {
        std::cerr << "CUDA is not available on this system." << std::endl;
    }

    return 0;
}
